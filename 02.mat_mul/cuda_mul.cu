#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "logger.hpp"
#include <stdio.h>

__global__ void MatmulKernel(float *dM1, float *dM2, float *dP) {

    int colIdx = blockIdx.x;
    int rowIdx = blockIdx.y;
    int width = gridDim.y;
    int k = threadIdx.x;

    atomicAdd(&dP[rowIdx * width + colIdx], dM1[rowIdx * width + k] * dM2[(rowIdx + k) * width + colIdx]);

//    printf("%d %d %d %d\n", colIdx, rowIdx, width, k);
}


void MatmulAtomicAddOnDevice(float *M_host, float *N_host, float *P_host, int width) {

    /* 设置矩阵大小 */
    int size = width * width * sizeof(float);

    /* 分配M, N在GPU上的空间*/
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc((void **) &M_device, size));
    CUDA_CHECK(hipMalloc((void **) &N_device, size));

    /* 分配M, N拷贝到GPU上*/
    CUDA_CHECK(hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice));

    /* 分配P在GPU上的空间*/
    float *P_device;
    CUDA_CHECK(hipMalloc((void **) &P_device, size));;

    dim3 dimGrid(width, width);
    dim3 dimBlock(width);
    MatmulKernel<<<dimGrid, dimBlock>>>(M_device, N_device, P_device);

    /* 将结果从device拷贝回host*/
    CUDA_CHECK(hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    /* 注意要在synchronization结束之后排查kernel的错误 */
    LAST_KERNEL_CHECK();

    /* Free */
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}



//--------------------------------share mem--------------------------------------//
#define BLOCKSIZE 16

/*
    使用shared memory把计算一个tile所需要的数据分块存储到访问速度快的memory中
*/
__global__ void MatmulSharedStaticKernel(float *M_device, float *N_device, float *P_device, int width) {
    __shared__ float M_deviceShared[BLOCKSIZE][BLOCKSIZE];
    __shared__ float N_deviceShared[BLOCKSIZE][BLOCKSIZE];
    /*
        对于x和y, 根据blockID, tile大小和threadID进行索引
    */
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_element = 0.0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    /* 对于每一个P的元素，我们只需要循环遍历width / tile_width 次就okay了，这里有点绕，画图理解一下*/
    for (int m = 0; m < width / BLOCKSIZE; m++) {
        M_deviceShared[ty][tx] = M_device[y * width + (m * BLOCKSIZE + tx)];
        N_deviceShared[ty][tx] = N_device[(m * BLOCKSIZE + ty) * width + x];
        __syncthreads();

        for (int k = 0; k < BLOCKSIZE; k++) {
            P_element += M_deviceShared[ty][k] * N_deviceShared[k][tx];
        }
        __syncthreads();
    }

    P_device[y * width + x] = P_element;
}

__global__ void MatmulSharedDynamicKernel(float *M_device, float *N_device, float *P_device, int width, int blockSize) {
    /*
        声明动态共享变量的时候需要加extern，同时需要是一维的
        注意这里有个坑, 不能够像这样定义：
            __shared__ float M_deviceShared[];
            __shared__ float N_deviceShared[];
        因为在cuda中定义动态共享变量的话，无论定义多少个他们的地址都是一样的。
        所以如果想要像上面这样使用的话，需要用两个指针分别指向shared memory的不同位置才行
    */

    extern __shared__ float deviceShared[];
    int stride = blockSize * blockSize;
    /*
        对于x和y, 根据blockID, tile大小和threadID进行索引
    */
    int x = blockIdx.x * blockSize + threadIdx.x;
    int y = blockIdx.y * blockSize + threadIdx.y;

    float P_element = 0.0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    /* 对于每一个P的元素，我们只需要循环遍历width / tile_width 次就okay了 */
    for (int m = 0; m < width / blockSize; m++) {
        deviceShared[ty * blockSize + tx] = M_device[y * width + (m * blockSize + tx)];
        deviceShared[stride + (ty * blockSize + tx)] = N_device[(m * blockSize + ty) * width + x];
        __syncthreads();

        for (int k = 0; k < blockSize; k++) {
            P_element += deviceShared[ty * blockSize + k] * deviceShared[stride + (k * blockSize + tx)];
        }
        __syncthreads();
    }

    if (y < width && x < width) {
        P_device[y * width + x] = P_element;
    }
}

/*
    使用Tiling技术
    一个tile处理的就是block, 将一个矩阵分为多个小的tile，这些tile之间的执行独立，并且可以并行
*/
void MatmulSharedOnDevice(float *M_host, float *N_host, float *P_host, int width, int blockSize, bool staticMem) {
    /* 设置矩阵大小 */
    int size = width * width * sizeof(float);
    long int sMemSize = blockSize * blockSize * sizeof(float) * 2;

    /* 分配M, N在GPU上的空间*/
    float *M_device;
    float *N_device;
    CUDA_CHECK(hipMalloc((void **) &M_device, size));
    CUDA_CHECK(hipMalloc((void **) &N_device, size));

    /* 分配M, N拷贝到GPU上*/
    CUDA_CHECK(hipMemcpy(M_device, M_host, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(N_device, N_host, size, hipMemcpyHostToDevice));

    /* 分配P在GPU上的空间*/
    float *P_device;
    CUDA_CHECK(hipMalloc((void **) &P_device, size));;

    /* 调用kernel来进行matmul计算, 在这个例子中我们用的方案是：使用一个grid，一个grid里有width*width个线程 */
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(width / blockSize, width / blockSize);
    if (staticMem) {
        MatmulSharedStaticKernel <<<dimGrid, dimBlock>>>(M_device, N_device, P_device, width);
    } else {
        MatmulSharedDynamicKernel <<<dimGrid, dimBlock, sMemSize, nullptr>>>(M_device, N_device, P_device, width, blockSize);
    }

    /* 将结果从device拷贝回host*/
    CUDA_CHECK(hipMemcpy(P_host, P_device, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    /* 注意要在synchronization结束之后排查kernel的错误 */
    LAST_KERNEL_CHECK();

    /* Free */
    hipFree(P_device);
    hipFree(N_device);
    hipFree(M_device);
}