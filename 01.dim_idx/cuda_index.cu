#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "logger.hpp"
#include <stdio.h>

__global__ void print_thread_idx_kernel() {
    int bSize = blockDim.z * blockDim.y * blockDim.x;

    int bIndex = blockIdx.z * gridDim.x * gridDim.y +
                 blockIdx.y * gridDim.x +
                 blockIdx.x;

    int tIndex = threadIdx.z * blockDim.x * blockDim.y +
                 threadIdx.y * blockDim.x +
                 threadIdx.x;

    int index = bIndex * bSize + tIndex;

    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n",
           bIndex, tIndex, index);
}


void print_idx_device(dim3 grid, dim3 block) {
    print_thread_idx_kernel<<<grid, block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
}
