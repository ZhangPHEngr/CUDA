#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "logger.hpp"

// #define MAX_ITER 10000     // memcpy == kernel / 10    (kernel执行的太快看不出来overlapping)
#define MAX_ITER 100000    // memcpy == kernel / 100   (开始能够看出来kernel的overlapping)
// #define MAX_ITER 10000000   // memcpy == kernel / 10000  (可以非常清楚的看到kernel的Overlapping)
#define SIZE 32

// 为了能够体现延迟，这里特意使用clock64()来进行模拟sleep
// 否则如果kernel计算太快，而无法观测到kernel在multi stream中的并发
// 大家根据自己的情况需修改sleep的时间
__global__ void SleepKernel(int64_t num_cycles) {
    int64_t cycles = 0;
    int64_t start = clock64();
    while (cycles < num_cycles) {
        cycles = clock64() - start;
    }
}

/* 1 stream，处理一次memcpy，以及n个kernel */
void SleepSingleStream(float *src_host, float *tar_host,
                       int width, int blockSize,
                       int count) {
    int size = width * width * sizeof(float);

    float *src_device;
    float *tar_device;

    CUDA_CHECK(hipMalloc((void **) &src_device, size));
    CUDA_CHECK(hipMalloc((void **) &tar_device, size));;


    for (int i = 0; i < count; i++) {
        for (int j = 0; j < 1; j++)
            CUDA_CHECK(hipMemcpy(src_device, src_host, size, hipMemcpyHostToDevice));

        dim3 dimBlock(blockSize, blockSize);
        dim3 dimGrid(width / blockSize, width / blockSize);

        SleepKernel <<<dimGrid, dimBlock >>>(MAX_ITER);
        CUDA_CHECK(hipMemcpy(src_host, src_device, size, hipMemcpyDeviceToHost));
    }


    CUDA_CHECK(hipDeviceSynchronize());


    hipFree(tar_device);
    hipFree(src_device);
}


/* n stream，处理一次memcpy，以及n个kernel */
void SleepMultiStream(float *src_host, float *tar_host,
                      int width, int blockSize,
                      int count) {
    int size = width * width * sizeof(float);

    float *src_device;
    float *tar_device;

    CUDA_CHECK(hipMalloc((void **) &src_device, size));
    CUDA_CHECK(hipMalloc((void **) &tar_device, size));


    /* 先把所需要的stream创建出来 */
    hipStream_t stream[count];
    for (int i = 0; i < count; i++) {
        CUDA_CHECK(hipStreamCreate(&stream[i]));
    }

    for (int i = 0; i < count; i++) {
        for (int j = 0; j < 1; j++)
            CUDA_CHECK(hipMemcpyAsync(src_device, src_host, size, hipMemcpyHostToDevice, stream[i]));
        dim3 dimBlock(blockSize, blockSize);
        dim3 dimGrid(width / blockSize, width / blockSize);

        /* 这里面我们把参数写全了 <<<dimGrid, dimBlock, sMemSize, stream>>> */
        SleepKernel <<<dimGrid, dimBlock, 0, stream[i]>>>(MAX_ITER);
        CUDA_CHECK(hipMemcpyAsync(src_host, src_device, size, hipMemcpyDeviceToHost, stream[i]));
    }


    CUDA_CHECK(hipDeviceSynchronize());


    hipFree(tar_device);
    hipFree(src_device);

    for (int i = 0; i < count; i++) {
        // 使用完了以后不要忘记释放
        hipStreamDestroy(stream[i]);
    }

}

/* n stream，处理一次memcpy，以及n个kernel */
